//
//
//                                                                                                                                          Asian_Test.h
//
//
// (c) Mark Joshi 2011
// This code is released under the GNU public licence version 3

/*

The purpose of this file is automate binding and unbinding of textures
*/


#include <cudaWrappers/cudaTextureBinder.h>



cudaTextureFloatBinder::cudaTextureFloatBinder(texture<float, 1, hipReadModeElementType>& texture_reference, thrust::device_ptr< float > dataPtr)
:
texture_reference_(texture_reference)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

    // set texture parameters
    texture_reference_.addressMode[0] = hipAddressModeWrap;
    texture_reference_.addressMode[1] = hipAddressModeWrap;
    texture_reference_.filterMode = hipFilterModeLinear;
    texture_reference_.normalized = false;    // access with normalized texture coordinates
    hipBindTexture( NULL, texture_reference_, thrust::raw_pointer_cast(dataPtr), channelDesc);

    std::cout << " device ptr float used\n";

}


cudaTextureFloatBinder::~cudaTextureFloatBinder()
{
    hipUnbindTexture(texture_reference_);
}

cudaTextureIntBinder::cudaTextureIntBinder(texture<int, 1>& texture_reference, 
                                           int* dataPtr)
                                           :
texture_reference_(texture_reference)
{

    hipBindTexture( NULL, texture_reference_, dataPtr);

}

cudaTextureIntBinder::cudaTextureIntBinder(texture<int, 1>& texture_reference, thrust::device_ptr< int > dataPtr)
:
texture_reference_(texture_reference)
{
    hipBindTexture( NULL, texture_reference_, thrust::raw_pointer_cast(dataPtr));
      std::cout << " device ptr int used\n";
}

cudaTextureIntBinder::~cudaTextureIntBinder()
{
    hipUnbindTexture(texture_reference_);
}
