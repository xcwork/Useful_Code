//
//
//                                                                                                                                          Asian_Test.h
//
//
// (c) Mark Joshi 2011
// This code is released under the GNU public licence version 3

/*

The purpose of this file is automate creation and deletion of cudaEvents
*/

#include <cudaWrappers/cudaEventWrapper.h>

cudaEventWrapper::cudaEventWrapper()
{
    cutilSafeCall(hipEventCreate(&inner_Event));
}

cudaEventWrapper::cudaEventWrapper(int flags)
{
    cutilSafeCall(hipEventCreate(&inner_Event,flags));
}

cudaEventWrapper::~cudaEventWrapper()
{
    cutilSafeCall(hipEventDestroy(inner_Event));
}


hipEvent_t& cudaEventWrapper::operator*()
{
    return inner_Event;
}

hipError_t cudaEventWrapper::query()
{
     return hipEventQuery(inner_Event);
}

hipError_t cudaEventWrapper::record()
{
    return hipEventRecord(inner_Event);
    
}

hipError_t cudaEventWrapper::record(hipStream_t stream)
{
    return hipEventRecord(inner_Event,stream);
    
}

hipError_t cudaEventWrapper::synchronize()
{
   return hipEventSynchronize(inner_Event);
}


std::pair<float, hipError_t > cudaEventWrapper::timeSince(cudaEventWrapper& startEvent)
{
    float t;
    hipError_t errVal( hipEventElapsedTime(&t, startEvent.inner_Event,inner_Event));

    return std::pair<float,hipError_t>(t,errVal);
}
