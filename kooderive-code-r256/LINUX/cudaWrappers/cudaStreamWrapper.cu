//
//
//                                                                                                                                          Asian_Test.h
//
//
// (c) Mark Joshi 2011
// This code is released under the GNU public licence version 3

/*

The purpose of this file is automate creation and deletion of cudaStreams
*/

#include <cudaWrappers/cudaStreamWrapper.h>

cudaStreamWrapper::cudaStreamWrapper()
{
    cutilSafeCall(hipStreamCreate(&inner_Stream));
}


cudaStreamWrapper::~cudaStreamWrapper()
{
    cutilSafeCall(hipStreamDestroy(inner_Stream));
}


hipStream_t& cudaStreamWrapper::operator*()
{
    return inner_Stream;
}

hipError_t cudaStreamWrapper::query()
{
     return hipStreamQuery(inner_Stream);
}



hipError_t cudaStreamWrapper::synchronize()
{
   return hipStreamSynchronize(inner_Stream);
}


